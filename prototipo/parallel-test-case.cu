#include "red-serial.hpp"
#include "large-case.hpp"
#include <iostream>
#include <vector>
#include <cassert>
#include "hipblas.h"
#include <omp.h>
#define N_TIME 1

typedef std::vector<float> vf;

void mult_mv(float *matrix,int matrix_rows,int matrix_cols,float *vector, int vector_size,float *result,int result_size){
        cublasInit();
        float *d_matrix;
        float *d_vector;
        float *d_result;

        // Asignación de memoria
        cublasAlloc(matrix_rows*matrix_cols,sizeof(float),(void**)&d_matrix);
        cublasAlloc(vector_size,sizeof(float),(void**)&d_vector);
        cublasAlloc(result_size,sizeof(float),(void**)&d_result);

        // Copiado de memoria Host to Device
        hipblasSetMatrix(matrix_rows,matrix_cols,sizeof(float),matrix,matrix_rows,d_matrix,matrix_rows);
        hipblasSetVector(vector_size,sizeof(float),vector,1,d_vector,1);

        // Llamada a kernel de cublas para multiplicar d_matrix por vector
        hipblasSgemv('t',matrix_cols,matrix_rows,1,d_matrix,matrix_cols,d_vector,1,0,d_result,1);
        hipDeviceSynchronize();

        hipblasGetVector(result_size,sizeof(float),d_result,1,result,1);

        cublasFree(d_matrix);
        cublasFree(d_vector);
        cublasFree(d_result);

        cublasShutdown();
}

void parallel_forward_pass(int n_input, int n_hidden, int n_output,
                    float input[], float hidden[], float output[],
                    float weights_ih[], float weights_ho[]){

    int h, j;
    const float hidden_bias = 1.0;
    const float output_bias = 1.0;

    /* Initialize hidden-layer neurons with zero. */
    std::fill_n(hidden, n_hidden, 0.0);

    // Procesamiento de capa escondida con CUDA-CUBLAS
    mult_mv(weights_ih,n_hidden,n_input,input,n_input,hidden,n_hidden);

    /* Hidden neuron activation */
    for (h = 0; h < n_hidden; ++h)
        hidden[h] = logistic(hidden[h] + hidden_bias);

    mult_mv(weights_ho,n_output,n_hidden,hidden,n_hidden,output,n_output);

    /* Output neuron activation */
    for (j = 0; j < n_output; ++j)
        output[j] = logistic(output[j] + output_bias);

}

int main() {
    vf input, hidden;
    vf computed_output, actual_output;
    vf weights_ih, weights_ho;
    float t_ini,t_end;

    std::cout << "Loading vector." << std::endl;
    load_matrix(input, INPUT_FILE);

    std::cout << "Loading weight_ih." << std::endl;
    load_matrix(weights_ih, WEIGHTS_IH_FILE);

    std::cout << "Loading weight_ho." << std::endl;
    load_matrix(weights_ho, WEIGHTS_HO_FILE);

    std::cout << "Loading actual output." << std::endl;
    load_matrix(actual_output, OUTPUT_FILE);

    assert(input.size() == INPUT_SIZE);
    assert(weights_ih.size() == INPUT_SIZE * HIDDEN_SIZE);
    assert(weights_ho.size() == HIDDEN_SIZE * OUTPUT_SIZE);
    assert(actual_output.size() == OUTPUT_SIZE);

    hidden.resize(HIDDEN_SIZE);
    computed_output.resize(OUTPUT_SIZE);

    std::cout << "Forward pass." << std::endl;
    t_ini = omp_get_wtime();
    for(int i = 0; i < N_TIME; i++){
    parallel_forward_pass(INPUT_SIZE, HIDDEN_SIZE, OUTPUT_SIZE,
            &input[0], &hidden[0], &computed_output[0],
            &weights_ih[0], &weights_ho[0]);
    }
    t_end = omp_get_wtime();


    std::cout << "First values of computed output: " << std::endl;
    print_matrix(computed_output, 1, 16);

    std::cout << "Actual first values: " << std::endl;
    print_matrix(actual_output, 1, 16);

    std::cout << "Forward Pass Time: " << std::endl;
    std::cout << (t_end - t_ini)/(float)N_TIME << std::endl;

}

