#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <vector>
#include <cstring>
#include <iterator>
#include "ibm.hpp"

/* fully-static version */
#define N 6
#define H 10
#define M 1

/* Threads per block */
#define T 32

__device__ float logistic(float x){
        return 1 / (1 + expf(-x));
}


__device__ float forward_pass
            (int n_input, int n_hidden, int n_output,
            float input[], float hidden[], float output[],
            float weights_ih[], float weights_ho[])
{
    int h, i, j;

    const float hidden_bias = 1.0;
    const float output_bias = 1.0;

    /* Initialize hidden-layer neurons with zero. */
    // std::fill_n(hidden, n_hidden, 0.0);
    for (int i = 0; i <  n_hidden; i++){
        hidden[i] = 0.0;
        output[i] = 0.0;
    }

    for (h = 0; h < n_hidden; ++h)
        for (i = 0; i < n_input; ++i)
            hidden[h] += weights_ih[h * n_input + i] * input[i];

    /* Hidden neuron activation */
    for (h = 0; h < n_hidden; ++h)
        hidden[h] = logistic(hidden[h] + hidden_bias);

    for (j = 0; j < n_output; ++j)
        for (h = 0; h < n_hidden; ++h)
            output[j] += weights_ho[j * n_hidden + h] * hidden[h];

    /* Output neuron activation */
    for (j = 0; j < n_output; ++j)
        output[j] = logistic(output[j] + output_bias);

    return output[0];
}

__global__ void cuda_forward_pass
            (int n_input, int n_hidden, int n_output,
            float *inputs, float *hiddens, float *outputs,
            float weights_ih[], float weights_ho[], float *tmp_hidden)
{
    int t = threadIdx.x;
    int b = blockIdx.x;

    __shared__ float sh_input[N * T];
    __shared__ float sh_hidden[H * T];
    __shared__ float sh_weights_ih[N * H];
    __shared__ float sh_weights_ho[H * M];

    float *start;
    int nr_iterations;

    /* fetch the input data, chunk by chunk */
    start = inputs + N * T * b;
    for (int i = 0; i < N; ++i)
        sh_input[t + i * T] = start[t + i * T];

    __syncthreads();

    /* fetch weights_ih, chunk by chunk */
    nr_iterations = (int) ceilf((N * H) / (float) T);
    for (int i = 0; i < nr_iterations; ++i) {
        int index = t + i * T;
        if (index < N * H)
            sh_weights_ih[index] = weights_ih[index];
    }

    __syncthreads();

    /* fetch weights_ho, chunk by chunk */
    nr_iterations = (int) ceilf(H * M / (float) T);
    for (int i = 0; i < nr_iterations; ++i) {
        int index = t + i * T;
        if (index < H * M)
            sh_weights_ho[index] = weights_ho[index];
    }

    __syncthreads();

    /* for the time being, the output is 1-dimensional */
    float *my_hidden = sh_hidden + t * H;
    float *my_input  = sh_input  + t * N;
    float *my_output = &outputs[t + b * T];
    __syncthreads();
    outputs[t + b * T] = forward_pass(n_input, n_hidden, n_output,
            my_input, my_hidden, my_output,
            sh_weights_ih, sh_weights_ho);
}



int main(){
    using namespace std;

    vector<daily_indicators> data;
    read_indicator_file(data, "../modelo-prueba/input.txt");

    vector<float> outputs, hiddens;
    vector<daily_indicators> inputs;
    inputs.reserve(data.size());
    outputs.reserve(data.size());
    hiddens.reserve(H*data.size());

    // CUDA Threads per Block
    static int TpB = 32;
    // CUDA blocks per Grid
    static int BpG = 32;

    float weights_ih[N * H];
    float weights_ho[H * M];

    read_weights_ih(weights_ih, N, H, "../modelo-prueba/weights_ih.txt");
    read_weights_ho(weights_ho, H, M, "../modelo-prueba/weights_ho.txt");

    float *d_inputs;
    float *d_outputs;
    float *d_weights_ih;
    float *d_weights_ho;
    float *d_hiddens;

    int n_data = data.size();

    hipMalloc((void**) &d_inputs,     n_data * N * sizeof(float));
    hipMalloc((void**) &d_outputs,    n_data * M * sizeof(float));
    hipMalloc((void**) &d_hiddens,    n_data * H * sizeof(float));

    hipMalloc((void**) &d_weights_ih, N * H * sizeof(float));
    hipMalloc((void**) &d_weights_ho, H * M * sizeof(float));


    hipMemcpy(d_inputs,     &data[0],   n_data * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights_ih, weights_ih, N * H * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights_ho, weights_ho, H * M * sizeof(float), hipMemcpyHostToDevice);

    cuda_forward_pass<<< BpG, TpB >>> (N, H, M,
                                         d_inputs, NULL, d_outputs,
                                         d_weights_ih, d_weights_ho, d_hiddens);

    hipDeviceSynchronize();

    hipMemcpy(&inputs[0],  d_inputs,  n_data * N * sizeof(float),     hipMemcpyDeviceToHost);
    hipMemcpy(&outputs[0], d_outputs, n_data * M * sizeof(float),     hipMemcpyDeviceToHost);
    hipMemcpy(&weights_ih[0], d_weights_ih, N * H  * sizeof(float),     hipMemcpyDeviceToHost);
    hipMemcpy(&weights_ho[0], d_weights_ho, H * M  * sizeof(float),     hipMemcpyDeviceToHost);
    hipMemcpy(&hiddens[0], d_hiddens,  H*T* sizeof(float),     hipMemcpyDeviceToHost);



    hipFree(d_inputs);
    hipFree(d_outputs);
    hipFree(d_hiddens);
    hipFree(d_weights_ih);
    hipFree(d_weights_ho);


    cout << "Output: " << endl;
    //copy(outputs.begin(), outputs.end(), ostream_iterator<float>(cout, "\n"));
    for (int i = 0; i < H; i++)
        cout << outputs[i] << endl;
    cout << endl;

    return 0;
}
