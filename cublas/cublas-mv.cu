#include <stdlib.h>
#include <stdio.h>
#include "hipblas.h"
#define HA 3
#define WA 3
#define S 3

void printMat(float*P,int uWP,int uHP){
    int i,j,z=0;

    for(i=0;i<uHP;i++){
        printf("\n");
        for(j=0;j<uWP;j++){
            printf("%f ",P[z]);
            z++;
        }
    }
}
void printVect(float*P,int sP){
    int j;

    for(j=0;j<sP;j++)
        printf("%f ",P[j]);

    printf("\n");
}

int  main (int argc, char** argv) {
    // Estructura para comprobar el estado de cualquier llamada cublas
    //hipblasStatus_t status;
    int i,j,z=0;
    cublasInit();

    float *A = (float*)malloc(HA*WA*sizeof(float));
    float *B = (float*)malloc(S*sizeof(float));
    float *C = (float*)malloc(S*sizeof(float));

    // llenado matriz A
    for (i=0;i<HA;i++){
        for (j=0;j<WA;j++){
            A[z] = (float) i+j+1;
            z++;
        }
    }

    // llenado vector B
    z = 0;
    for (i=0;i<S;i++){
       B[z] = i+1;
       C[z] = 0;
        z++;
    }

    float* AA;
    float* BB;
    float* CC;

    // Asignación de memoria
    cublasAlloc(HA*WA,sizeof(float),(void**)&AA);
    cublasAlloc(S,sizeof(float),(void**)&BB);
    cublasAlloc(S,sizeof(float),(void**)&CC);

    // Copiado de memoria Host to Device
    hipblasSetMatrix(HA,WA,sizeof(float),A,HA,AA,HA);
    hipblasSetVector(S,sizeof(float),B,1,BB,1);

    // Llamada a kernel de cublas para multiplicar matrix por vector
    hipblasSgemv('t',HA,WA,1,AA,WA,BB,1,0,CC,1);
    hipDeviceSynchronize();

    // Copiado de memoria Device to Host
    hipblasGetVector(S,sizeof(float),CC,1,C,1);

    // Verificacion contenido
    printf("\nMatriz A:\n");
    printMat(A,WA,HA);
    printf("\nVector B:\n");
    printVect(B,S);
    printf("\nVector C:\n");
    printVect(C,S);

    free(A);
    free(B);
    free(C);
    cublasFree(AA);
    cublasFree(BB);
    cublasFree(CC);

    cublasShutdown();
    return EXIT_SUCCESS;
  }
