#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "hipblas.h"
#include <iostream>
#define HA 4
#define WA 3
#define S 3
using namespace std;

void printMat(float*P,int uWP,int uHP){
    int i,j,z=0;

    for(i=0;i<uHP;i++){
        printf("\n");
        for(j=0;j<uWP;j++){
            printf("%f ",P[z]);
            z++;
        }
  }
}
void printVect(float*P,int sP){
    int j;

    for(j=0;j<sP;j++)
        printf("%f ",P[j]);

    printf("\n");
}

__global__ void smv(float A[], float B[], float C[], int height,int s)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    __shared__ extern float vec_s[];
    if(threadIdx.x < s)
        vec_s[threadIdx.x] = B[threadIdx.x];
    __syncthreads();

    if(tid < height){
      #pragma unroll 8
      for(int i=0; i < s; i++){
          sum += A[tid*s+i] * vec_s[i];
      }
      C[tid] = sum;
    }
}


int  main (int argc, char** argv) {
    int i,j,z=0;
    int TpB = 32;
    int BpG = 32;

    float *A = (float*)malloc(HA*WA*sizeof(float));
    float *B = (float*)malloc(S*sizeof(float));
    float *C = (float*)malloc(HA*sizeof(float));

    // llenado matriz A
    for (i=0;i<HA;i++){
        for (j=0;j<WA;j++){
            A[z] = (float) i+j+1;
            z++;
        }
    }
    cout << z << endl;
    // llenado vector B
    z = 0;
    for (i=0;i<S;i++){
       B[z] = i+1;
       C[z] = 0;
        z++;
    }

    z = 0;
    for (i=0;i<HA;i++){
       C[z] = 0;
        z++;
    }

    float* AA;
    float* BB;
    float* CC;

    hipMalloc((void**)&AA,HA*WA*sizeof(float));
    hipMalloc((void**)&BB,S*sizeof(float));
    hipMalloc((void**)&CC,HA*sizeof(float));

    hipMemcpy(AA , A , HA*WA*sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(BB , B , S*sizeof(float)     , hipMemcpyHostToDevice);

    smv <<< BpG, TpB, S >>> (AA,BB,CC,HA,S);
    hipDeviceSynchronize();

    hipMemcpy(C,CC,HA*sizeof(float),hipMemcpyDeviceToHost);

    // Verificacion contenido
    printf("\nMatriz A:\n");
    printMat(A,WA,HA);
    printf("\nVector B:\n");
    printVect(B,S);
    printf("\nVector C:\n");
    printVect(C,HA);

    free(A);
    free(B);
    free(C);
    hipFree(AA);
    hipFree(BB);
    hipFree(CC);
  }
