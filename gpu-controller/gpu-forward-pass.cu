#include "hip/hip_runtime.h"
#include "gpu-forward-pass.hpp"
#include "hipblas.h"


__global__ void set_one(float *array, int i) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id == 0) {
        array[i] = 1.0f;
    }
}


__device__ float gpu_logistic(float x) {
    return 1 / (1 + expf(-x));
}


__global__ void activation_function(float x[], int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < n){
        x[id] = gpu_logistic(x[id]);
    }
}


void initialize_gpu_arrays(int n_input,  float d_input[],
                           int n_hidden, float d_hidden[],
                           int n_output, float d_output[]) {
    set_one<<<1, 1>>>(d_input,  n_input);
    set_one<<<1, 1>>>(d_hidden, n_hidden);
}


/* Matrix-vector multiplication b = Ax
 * for a contiguous, row-major matrix A.
 * Matrix A has to have been set like this:
 *
 *   st = hipblasSetMatrix(COLS, ROWS, _, a_host, COLS, a_device, COLS);
 */
void matrix_vector_product(int nr_rows, int nr_cols,
                           float *b, float *a, float *x) {
    hipblasSgemv('t', nr_cols, nr_rows,
      1.0f, a, nr_cols,
            x, 1,
      0.0f, b, 1);
}


void gpu_forward_pass(int n_input, int n_hidden, int n_output,
                      float input[], float hidden[], float output[],
                      float d_input[], float d_hidden[], float d_output[],
                      float d_weights_ih[], float d_weights_ho[],
                      int activation)
{
    /* Assumption: d_input and d_hidden already have their ones at their tails.
     * (initialize_gpu_arrays must have already been called).  */

    int T, B;

    hipblasSetVector(n_input, sizeof(float), input, 1, d_input, 1);

    matrix_vector_product(
      /* matrix size: */ n_hidden, n_input + 1,
      /* operands:    */ d_hidden, d_weights_ih, d_input);

    T = 128;
    B = int(std::ceil(n_hidden / float(T)));
    activation_function<<<B, T>>>(d_hidden, n_hidden);

    hipDeviceSynchronize();

    matrix_vector_product(
      /* matrix size: */ n_output, n_hidden + 1,
      /* operands:    */ d_output, d_weights_ho, d_hidden);

    if (activation) {
        T = 128;
        B = int(std::ceil(n_hidden / float(T)));
        activation_function<<<B, T>>>(d_output, n_output);
        hipDeviceSynchronize();
    }

    hipblasGetVector(n_output, sizeof(float), d_output, 1, output, 1);

}

